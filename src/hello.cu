#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main(int argc, char **argv) {
    printf("Hello World from CPU!\n");
    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize();
    return 0;
}
